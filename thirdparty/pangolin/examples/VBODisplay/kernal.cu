
#include <hip/hip_runtime.h>
// Colour Sine wave Kernal
// Based on kernal_colour in kernelVBO.cpp by Rob Farber
__global__ void kernel(float4* dVertexArray, uchar4 *dColorArray,
           unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // Each thread is unique point (u,v) in interval [-1,1],[-1,1]
    const float u = 2.0f* (x/(float)width)  - 1.0f;
    const float v = 2.0f* (y/(float)height) - 1.0f;
    const float w = 0.5f * sinf(4.0f*u + time) * cosf(4.0f*v + time);

    // Update vertex array for point
    dVertexArray[y*width+x] = make_float4(u, w, v, 1.0f);

    // Update colour array for point
    dColorArray[y*width+x].w = 0.0f;
    dColorArray[y*width+x].x = 255.0f *0.5f*(1.f+sinf(w+x));
    dColorArray[y*width+x].y = 255.0f *0.5f*(1.f+sinf(x)*cosf(y));
    dColorArray[y*width+x].z = 255.0f *0.5f*(1.f+sinf(w+time/10.0f));
}

extern "C" void launch_kernel(float4* dVertexArray, uchar4* dColourArray,
            unsigned int width, unsigned int height, float time)
{
  dim3 block(8, 8, 1);
  dim3 grid(width / block.x, height / block.y, 1);
  kernel<<< grid, block>>>(dVertexArray, dColourArray, width, height, time);
}
